// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>

#include <math.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	//c[threadIdx.x] = threadIdx.x;
	for (int i = 0; i < N; i++)
		c[i] = sqrt(c[i]);
}

int main()
{
	float *c = new float[N];	
	float *cd;
	const int size = N*sizeof(float);
	
	//cd = malloc(size);

	printf("Part 1\n");
	for (int i = 0; i < N; i++){
		c[i] = rand();
		printf("%f ", sqrt(c[i]));
	}
		

	printf("\n");

	printf("Part 2\n");

	hipMalloc( (void**)&cd, size );
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hipMemcpy( cd, c, size, hipMemcpyHostToDevice ); 
	simple<<<dimGrid, dimBlock>>>(cd);

	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	for (int i = 0; i < N; i++)
		printf("%f ", c[i]);
		
	hipFree( cd );
	

	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
