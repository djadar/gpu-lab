// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>

#include <math.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	c[threadIdx.x] = threadIdx.x;
}

int main()
{
	float *c = new float[N];	
	float *cd;
	const int size = N*sizeof(float);
	
	//cd = malloc(size);

	printf("Part 1\n");
	for (int i = 0; i < N; i++){
		c[i] = rand();
		printf("%f ", sqrt(c[i]));
	}
		

	printf("\n");

	printf("Part 2\n");

	hipMalloc( (void**)&cd, size );
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( cd, c, size, hipMemcpyHostToDevice ); 
	for (int i = 0; i < N; i++)
		printf("%f ", sqrt(c[i]));
		
	hipFree( cd );
	

	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
