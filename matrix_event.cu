// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>

#include <math.h>

#include <chrono>

using namespace std::chrono;

auto start = high_resolution_clock::now();

const int N = 16; 
const int blocksize = 16; 

__global__ 
void add_matrix(float *a, float *b, float *c) 
{
	//calculation of block index
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	c[index] = a[index] + b[index];
}

int main()
{
	float *a = new float[N*N];	
	float *b = new float[N*N];	
	float *c = new float[N*N];	

	float *ca, *cb, *cc;
	const int size = N*N*sizeof(float);
	
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = float(j) / N;
		}

	//GPU variables
	hipMalloc( (void**)&ca, size );
	hipMalloc( (void**)&cb, size );
	hipMalloc( (void**)&cc, size );

	//Block size
	dim3 dimBlock( blocksize, 1);
	dim3 dimGrid( 1, 1 );
	
	
	//Copy of CPU variables to GPU variables 
	hipMemcpy( ca, a, size, hipMemcpyHostToDevice ); 
	hipMemcpy( cb, b, size, hipMemcpyHostToDevice ); 

	//
	add_matrix<<<dimGrid, dimBlock>>>(ca,cb,cc);
	//

	hipDeviceSynchronize();
	
	//Copy of results from GPU variables to CPU variables
	hipMemcpy( c, cc, size, hipMemcpyDeviceToHost ); 

	//printing result
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%f",c[i+j*N]);
			printf("\n");
		}
	}
	printf("\n");

	//Free GPU memory
	hipFree( ca );
	hipFree( cb );
	hipFree( cc );

	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
